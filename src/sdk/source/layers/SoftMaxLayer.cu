#include "hip/hip_runtime.h"
#include <layers/SoftMaxLayer.hpp>

#include <hip/hip_runtime.h>

#include <stdexcept>
#include <iostream>

namespace
{

const unsigned int THREADS_PER_BLOCK = 1024;

// Utils
__device__
int64_t
countDim(const Dim *inDim, int64_t inAxis = 4)
{
    int64_t theCount = 0;
    if (inAxis >= 1)
        theCount = inDim->x;
    if (inAxis >= 2)
        theCount *= inDim->y;
    if (inAxis >= 3)
        theCount *= inDim->z;
    if (inAxis >= 4)
        theCount *= inDim->w;
    return theCount;
}

__device__
int64_t
indexToOffset(int64_t inLinearIndex, const Dim *inDim, const Dim *inStrides)
{
    int64_t theXCount = countDim(inDim, 1);
    int64_t theYCount = countDim(inDim, 2);
    int64_t theZCount = countDim(inDim, 3);
    int64_t w = inLinearIndex / theZCount;
    inLinearIndex -= theZCount * w;
    int64_t z = inLinearIndex / theYCount;
    inLinearIndex -= theYCount * z;
    int64_t y = inLinearIndex / theXCount;
    inLinearIndex -= theXCount * y;
    int64_t x = inLinearIndex;
    return inStrides->w * w + inStrides->z * z + inStrides->y * y + inStrides->x * x;
}

// Get max
template<typename TDataType>
__device__
TDataType
getMaxOfTwo(TDataType inArg1, TDataType inArg2)
{
    return inArg1 >= inArg2 ? inArg1 : inArg2;
}

template<typename TDataType>
__device__
inline
TDataType
ShuffleDown(TDataType inData, unsigned int inOffset)
{
    return __shfl_down(inData, inOffset);
}

template<>
__device__
inline
double
ShuffleDown<double>(double inData, unsigned int inOffset)
{
    int2 theDataForShuffle = *reinterpret_cast<int2*>(&inData);
    int2 theShuffled;
    theShuffled.x = __shfl_down(theDataForShuffle.x, inOffset);
    theShuffled.y = __shfl_down(theDataForShuffle.y, inOffset);
    return *reinterpret_cast<double*>(&theShuffled);
}

template<typename TDataType>
__device__
inline
TDataType
calExp(TDataType inOprand)
{
    return expf(inOprand);
}

template<>
__device__
inline
double
calExp<double>(double inOprand)
{
    return exp(inOprand);
}

//TODO[sjiagc]: Refactor to remove duplicated code

template<typename TDataType>
__global__
void
getMax(TDataType *outMaxes, const TDataType *inData, Dim inDim, Dim inStrides)
{
    extern __shared__ TDataType theMaxes_shared[];
    __shared__ int64_t theElementCount;
    unsigned int theThreadId = threadIdx.x;
    unsigned int theGridSize = THREADS_PER_BLOCK * gridDim.x * 2;

    if (theThreadId == 0)
        theElementCount = countDim(&inDim);
    __syncthreads();

    TDataType  theLocalMax = inData[0];

    int64_t theIndexOfThread = blockIdx.x * THREADS_PER_BLOCK * 2 + theThreadId;
    while (theIndexOfThread < theElementCount) {
        theLocalMax = getMaxOfTwo(theLocalMax, inData[indexToOffset(theIndexOfThread, &inDim, &inStrides)]);
        if (theIndexOfThread + THREADS_PER_BLOCK < theElementCount)
            theLocalMax = getMaxOfTwo(theLocalMax, inData[indexToOffset(theIndexOfThread + THREADS_PER_BLOCK, &inDim, &inStrides)]);
        theIndexOfThread += theGridSize;
    }

    theMaxes_shared[theThreadId] = theLocalMax;
    __syncthreads();

    if (THREADS_PER_BLOCK >= 1024 && theThreadId < 512) {
        theMaxes_shared[theThreadId] = theLocalMax = getMaxOfTwo(theLocalMax, theMaxes_shared[theThreadId + 512]);
    }
    __syncthreads();
    if (THREADS_PER_BLOCK >= 512 && theThreadId < 256) {
        theMaxes_shared[theThreadId] = theLocalMax = getMaxOfTwo(theLocalMax, theMaxes_shared[theThreadId + 256]);
    }
    __syncthreads();
    if (THREADS_PER_BLOCK >= 256 && theThreadId < 128) {
        theMaxes_shared[theThreadId] = theLocalMax = getMaxOfTwo(theLocalMax, theMaxes_shared[theThreadId + 128]);
    }
    __syncthreads();
    if (THREADS_PER_BLOCK >= 128 && theThreadId < 64) {
        theMaxes_shared[theThreadId] = theLocalMax = getMaxOfTwo(theLocalMax, theMaxes_shared[theThreadId + 64]);
    }
    __syncthreads();

    if (theThreadId < 32) {
        if (THREADS_PER_BLOCK >= 64)
            theLocalMax = getMaxOfTwo(theLocalMax, theMaxes_shared[theThreadId + 32]);
#pragma unroll
        for (unsigned int theOffset = warpSize / 2; theOffset > 0; theOffset /= 2) {
            TDataType theShuffled = ShuffleDown(theLocalMax, theOffset);
            theLocalMax = getMaxOfTwo(theLocalMax, theShuffled);
        }
    }
    if (theThreadId == 0)
        outMaxes[theThreadId] = theLocalMax;
}

template<typename TDataType>
__global__
void
expAll(TDataType *outOutput, Dim inDim, Dim inOutputStrides, const TDataType *inInput, Dim inInputStrides, TDataType *inMax)
{
    unsigned int theThreadId = threadIdx.x;
    int64_t theIndexOfThread = blockIdx.x * THREADS_PER_BLOCK + theThreadId;
    int64_t theOutputOffset = indexToOffset(theIndexOfThread, &inDim, &inOutputStrides);
    int64_t theInputOffset = indexToOffset(theIndexOfThread, &inDim, &inInputStrides);
    outOutput[theOutputOffset] = calExp<TDataType>(inInput[theInputOffset] - *inMax);
}

template<typename TDataType>
__global__
void
sumUp(TDataType *outSum, const TDataType *inData, Dim inDim, Dim inStrides)
{
    extern __shared__ TDataType theSums_shared[];
    __shared__ int64_t theElementCount;
    unsigned int theThreadId = threadIdx.x;
    unsigned int theGridSize = THREADS_PER_BLOCK * gridDim.x * 2;

    if (theThreadId == 0)
        theElementCount = countDim(&inDim);
    __syncthreads();

    TDataType  theLocalSum = 0;

    int64_t theIndexOfThread = blockIdx.x * THREADS_PER_BLOCK * 2 + theThreadId;
    while (theIndexOfThread < theElementCount) {
        theLocalSum += inData[indexToOffset(theIndexOfThread, &inDim, &inStrides)];
        if (theIndexOfThread + THREADS_PER_BLOCK < theElementCount)
            theLocalSum += inData[indexToOffset(theIndexOfThread + THREADS_PER_BLOCK, &inDim, &inStrides)];
        theIndexOfThread += theGridSize;
    }

    theSums_shared[theThreadId] = theLocalSum;
    __syncthreads();

    if (THREADS_PER_BLOCK >= 1024 && theThreadId < 512) {
        theSums_shared[theThreadId] = theLocalSum = theLocalSum + theSums_shared[theThreadId + 512];
    }
    __syncthreads();
    if (THREADS_PER_BLOCK >= 512 && theThreadId < 256) {
        theSums_shared[theThreadId] = theLocalSum = theLocalSum + theSums_shared[theThreadId + 256];
    }
    __syncthreads();
    if (THREADS_PER_BLOCK >= 256 && theThreadId < 128) {
        theSums_shared[theThreadId] = theLocalSum = theLocalSum + theSums_shared[theThreadId + 128];
    }
    __syncthreads();
    if (THREADS_PER_BLOCK >= 128 && theThreadId < 64) {
        theSums_shared[theThreadId] = theLocalSum = theLocalSum + theSums_shared[theThreadId + 64];
    }
    __syncthreads();

    if (theThreadId < 32) {
        if (THREADS_PER_BLOCK >= 64)
            theLocalSum += theSums_shared[theThreadId + 32];
#pragma unroll
        for (unsigned int theOffset = warpSize / 2; theOffset > 0; theOffset /= 2) {
            TDataType theShuffled = ShuffleDown(theLocalSum, theOffset);
            theLocalSum += theShuffled;
        }
    }
    if (theThreadId == 0)
        outSum[theThreadId] = theLocalSum;
}

template<typename TDataType>
__global__
void
calProb(TDataType *outOutput, Dim inDim, Dim inOutputStrides, const TDataType *inInput, Dim inInputStrides, TDataType *inSum)
{
    unsigned int theThreadId = threadIdx.x;
    int64_t theIndexOfThread = blockIdx.x * THREADS_PER_BLOCK + theThreadId;
    int64_t theOutputOffset = indexToOffset(theIndexOfThread, &inDim, &inOutputStrides);
    int64_t theInputOffset = indexToOffset(theIndexOfThread, &inDim, &inInputStrides);
    outOutput[theOutputOffset] = inInput[theInputOffset] / *inSum;
}

}

namespace layer
{

template<typename TDataType>
void
SoftMaxLayer<TDataType>::forwardGPU()
{
    const utils::Dimension &theInputDim = m_input->getDimension();
    const utils::Dimension &theInputStrides = m_input->getStride();
    const utils::Dimension &theOutputDim = m_data->getDimension();
    const utils::Dimension &theOutputStrides = m_data->getStride();
    int64_t theElementCount = theOutputDim.count();

    unsigned int theBlockCount = theElementCount / THREADS_PER_BLOCK > 0 ? theElementCount / THREADS_PER_BLOCK : 1;
    TDataType *theIntermediateBuffer_d = nullptr;
    hipMalloc(&theIntermediateBuffer_d, sizeof(TDataType) * theBlockCount);
    // Get max
    {
        TDataType *theOutput_d = theIntermediateBuffer_d;
        const TDataType *theInput_d = m_input->getGPUData();
        unsigned int theResultCount = theBlockCount;
        Dim theDim = theInputDim.toDim();
        Dim theStrides = theInputStrides.toDim();
        hipError_t theCudaStatus = hipSuccess;
        while (theResultCount) {
            getMax<TDataType><<<theBlockCount, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(TDataType)>>>(theOutput_d, theInput_d, theDim, theStrides);
            theCudaStatus = hipGetLastError();
            if (theCudaStatus != hipSuccess)
                throw std::runtime_error("SoftMaxLayer::forwardGPU: executing kernel for max failed");
            if (theResultCount <= 1)
                break;
            theDim.w = theDim.z = theDim.y = 1;
            theDim.x = theResultCount;
            theStrides.w = theStrides.z = theStrides.y = theResultCount;
            theStrides.x = 1;
            theResultCount = theResultCount/ THREADS_PER_BLOCK > 0 ? theResultCount/ THREADS_PER_BLOCK : 1;
        }
//        TDataType theMax = 0;
//        theCudaStatus = hipMemcpy(&theMax, theOutput_d, sizeof(TDataType), hipMemcpyDeviceToHost);
//        if (theCudaStatus != hipSuccess) {
//            throw std::runtime_error(std::string("SoftMaxLayer::forwardGPU: get max failed, ") + hipGetErrorString(theCudaStatus));
//        }
//        std::cout << "Max: " << theMax << std::endl;
    }
    // Calculate exp
    {
        expAll<TDataType><<<(theElementCount + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>
            (m_data->getMutableGPUData(), theOutputDim.toDim(), theOutputStrides.toDim(), m_input->getGPUData(), theInputStrides.toDim(), theIntermediateBuffer_d);
    }
    // Sum
    {
        TDataType *theOutput_d = theIntermediateBuffer_d;
        const TDataType *theInput_d = m_data->getGPUData();
        unsigned int theResultCount = theBlockCount;
        Dim theDim = theOutputDim.toDim();
        Dim theStrides = theOutputStrides.toDim();
        hipError_t theCudaStatus = hipSuccess;
        while (theResultCount) {
            sumUp<TDataType><<<theBlockCount, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(TDataType)>>>(theOutput_d, theInput_d, theDim, theStrides);
            theCudaStatus = hipGetLastError();
            if (theCudaStatus != hipSuccess)
                throw std::runtime_error("SoftMaxLayer::forwardGPU: executing kernel for max failed");
            if (theResultCount <= 1)
                break;
            theDim.w = theDim.z = theDim.y = 1;
            theDim.x = theResultCount;
            theStrides.w = theStrides.z = theStrides.y = theResultCount;
            theStrides.x = 1;
            theResultCount = theResultCount/ THREADS_PER_BLOCK > 0 ? theResultCount/ THREADS_PER_BLOCK : 1;
        }
//        TDataType theSum = 0;
//        theCudaStatus = hipMemcpy(&theSum, theOutput_d, sizeof(TDataType), hipMemcpyDeviceToHost);
//        if (theCudaStatus != hipSuccess) {
//            throw std::runtime_error(std::string("SoftMaxLayer::forwardGPU: get max failed, ") + hipGetErrorString(theCudaStatus));
//        }
//        std::cout << "Sum: " << theSum << std::endl;
    }
    // Calculate probability
    {
        calProb<TDataType><<<(theElementCount + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>
            (m_data->getMutableGPUData(), theOutputDim.toDim(), theOutputStrides.toDim(), m_data->getGPUData(), theOutputStrides.toDim(), theIntermediateBuffer_d);
    }

    hipFree(theIntermediateBuffer_d);
}

template void SoftMaxLayer<double>::forwardGPU();

}
