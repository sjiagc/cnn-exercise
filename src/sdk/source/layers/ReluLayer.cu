#include "hip/hip_runtime.h"
#include <layers/ReluLayer.hpp>

#include <stdexcept>

namespace
{

const unsigned int BLOCK_DIM = 1024;

template<typename TDataType>
__global__
void
ReluLayer_forwardKernel(TDataType *inDst, const TDataType *inSrc, int64_t inElementCount, TDataType inNegativeSlope)
{
    int theThreadId = blockIdx.x * gridDim.x + threadIdx.x;
    if (inElementCount <= theThreadId)
        return;
    if (inSrc[theThreadId] <= 0)
        inDst[theThreadId] = inSrc[theThreadId] * inNegativeSlope;
    else
        inDst[theThreadId] = inSrc[theThreadId];
}

}

namespace layer
{

template<typename TDataType>
void
ReluLayer<TDataType>::forwardGPU()
{
    struct Dim theDim = m_input->getDimension().toDim();
    int64_t theElementCount = theDim.w * theDim.z * theDim.y * theDim.x;
    ReluLayer_forwardKernel<TDataType><<<static_cast<unsigned int>((theElementCount + BLOCK_DIM - 1) / BLOCK_DIM), BLOCK_DIM>>>
        (m_data->getMutableGPUData(), m_input->getGPUData(), theElementCount, m_negativeSlope);
    if (hipGetLastError() != hipSuccess)
        throw std::runtime_error("ReluLayer::forwardGPU: kernel execution failed");
}

template void ReluLayer<double>::forwardGPU();

}
